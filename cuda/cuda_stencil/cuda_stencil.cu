
#include <hip/hip_runtime.h>
#include <iostream>
#include <stdio.h>

#define N  100
#define ITERS 5

__global__ void stencil(float* a, float* b) {
  int x = blockIdx.x;
  int y = blockIdx.y;
  int offset = x + y * N;

  float update = 0.0;
  if (y > 0) {
    update += a[(y-1)*N+x];
  }
  if (y < N-1) {
    update += a[(y+1)*N+x];
  }
  if (x > 0) {
    update += a[y*N+(x-1)];
  }
  if (x < N-1) {
    update += a[y*N+(x+1)];
  }

  b[offset] = update / 4.0;
}

__global__ void copy(float* to, float* from) {
  int offset = blockIdx.x + blockIdx.y * N;
  to[offset] = from[offset];
}

int main() {
  float a[N*N], b[N*N];
  float *dev_a, *dev_b;

  dim3 blocks(N, N);

  hipMalloc((void**)&dev_a, N*N*sizeof(float));
  hipMalloc((void**)&dev_b, N*N*sizeof(float));

  for (int i = 0; i < N; i++) {
    for (int j = 0; j < N; j++) {
      a[i*N+j] = static_cast<float>(i+j);
    }
  }

  hipMemcpy(dev_a, a, N*N*sizeof(float), hipMemcpyHostToDevice);
  hipMemcpy(dev_b, b, N*N*sizeof(float), hipMemcpyHostToDevice);

  for (int num_it = 0; num_it < ITERS; num_it++) {
    stencil<<<blocks, 1>>>(dev_a, dev_b);
    copy<<<blocks, 1>>>(dev_a, dev_b);
  }

  hipMemcpy(b, dev_b, N*N*sizeof(float), hipMemcpyDeviceToHost);

  // print out the new array b
  std::cout << std::endl;
  for (int i = 0; i < N; i++) {
    for (int j = 0; j < N; j++) {
    }
  }
  std::cout << std::endl;

  // find sum
  float sum = 0.0;
  for (int i = 0; i < N; i++) {
    for (int j = 0; j < N; j++) {
      sum += b[i*N+j];
      std::cout << b[i*N+j] << " ";
    }
    std::cout << std::endl;
  }
  std::cout << "sum is " << sum << std::endl;
  hipFree(dev_a);
  hipFree(dev_b);
}


